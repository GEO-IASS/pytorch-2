
#include <hip/hip_runtime.h>
#ifndef THCS_GENERIC_FILE
#define THCS_GENERIC_FILE "generic/THCSTensor.cu"
#else

THCTensor *THCSTensor_(toDense)(THCState *state, THCSTensor *self) {
  /*
  THLongStorage *storage;
  THCTensor *other;

  THCSTensor_(contiguous)(state, self);

  // set up the new tensor
  storage = THCSTensor_(newSizeOf)(state, self);
  other = THCTensor_(newWithSize)(state, storage, NULL);
  THCTensor_(zero)(state, other);

  // Some necessary dimensions and sizes
  const ptrdiff_t nnz = THCSTensor_(nnz)(state, self);
  const dim3 block = getApplyBlock();
  dim3 grid;
  THArgCheck(getApplyGrid(state, nnz, grid), 1, CUTORCH_DIM_WARNING);

  TensorInfo<real, uint64_t> otherInfo =
    getTensorInfo<THCTensor, uint64_t>(state, other);
  TensorInfo<int64_t, uint64_t> indicesInfo =
    getTensorInfo<THCudaLongTensor, uint64_t>(state, self->indices);
  TensorInfo<real, uint64_t> valuesInfo =
    getTensorInfo<THCTensor, uint64_t>(state, self->values);

  THCSTensor_toDenseKernel<uint64_t, real>
    <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
        otherInfo, indicesInfo, valuesInfo, (uint64_t)(nnz));

  THCudaCheck(cudaGetLastError());
  THLongStorage_free(storage);
  return other;
  */
  THError("WARNING: Sparse Cuda Tensor op toDense is not implemented");
  return NULL;
}

void THCSTensor_(reorder)(THCState *state, THCSTensor *self) {
  THError("WARNING: Sparse Cuda Tensor op reorder is not implemented");
}

void THCSTensor_(contiguous)(THCState *state, THCSTensor *self) {
  if (self->contiguous) return;
  THCSTensor_(reorder)(state, self);
  self->contiguous = 1;
}

// In place transpose
void THCSTensor_(transpose)(THCState *state, THCSTensor *self, int d1, int d2) {
  /* TODO
  THCudaLongTensor *indices = THCSTensor_(indices)(state, self);
  int64_t i;
  for (i = 0; i < THCSTensor_(nnz)(state, self); i++) {
    int64_t tmp = THCTensor_fastGet2d(indices, d1, i);
    THCTensor_fastSet2d(indices, d1, i,
        THCTensor_fastGet2d(indices, d2, i));
    THCTensor_fastSet2d(indices, d2, i, tmp);
  }
  i = self->size[d1];
  self->size[d1] = self->size[d2];
  self->size[d2] = i;
  self->contiguous = 0;
  THFree(indices);
  */
  THError("WARNING: Sparse Cuda Tensor op transpose is not implemented");
}

int THCSTensor_(getDevice)(THCState* state, const THCSTensor* tensor) {
  if (!tensor->values || !tensor->values->storage) return -1;
  return THCStorage_(getDevice)(state, tensor->values->storage);
}

void THCTensor_(sparseMask)(THCState *state, THCSTensor *r_, THCTensor *t, THCSTensor *mask) {
  THError("WARNING: Sparse Cuda Tensor op sparseMask is not implemented");
}

#endif
